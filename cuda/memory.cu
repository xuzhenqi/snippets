#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

int main(int argc, char* argv[]) {
  long long size = 0;
  if (argc < 2) {
    size = 2000000000;
  } else {
    size = stoll(argv[1]) * 1000000;
  }
  cout << "size: " << size << endl;
  float *pd[10];
  hipError_t err = hipSuccess;
  for (int i = 0; i < 10; ++i) {
    err = hipMalloc(pd+i, size/10);
    if (err != hipSuccess) {
      cout << i << "hipMalloc Failed" << endl;
    }
  }
  cout << "Type a key: ";
  cin.ignore();
  for (int i = 0; i < 10; ++i) {
    hipFree(pd + i);
  }
  return 0;
}
